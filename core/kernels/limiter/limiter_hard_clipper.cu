
#include <hip/hip_runtime.h>
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

extern "C" __global__
void limiter_kernel(
    const double* __restrict__ wav_in, // Input audio signal [L0, R0, L1, R1, ..., LN-1, RN-1]
    double* __restrict__ wav_out,      // Output audio signal
    const int n_channels,       // Number of channels (e.g., 2 for stereo)
    const int n_samples,        // Number of samples per channel
    const double threshold,     // Threshold in linear scale (e.g., 0.5 for 50%)
    const double slope,         // Slope parameter for gain calculation
    const double sr,            // Sample rate in Hz
    const double tatt,          // Attack time in ms
    const double trel           // Release time in ms
)
{
    int ch = blockIdx.x;  // Each block processes one channel
    int thread_id = threadIdx.x; // Thread within the block
    int stride = blockDim.x;     // Number of threads in the block

    if (ch >= n_channels) return;

    int start_idx = thread_id * (n_samples / stride);
    int end_idx = (thread_id + 1) * (n_samples / stride);

    double clip_limit = threshold;
    double clip_limit_inv = 1.0 / clip_limit;

    for (int i = start_idx; i < end_idx; ++i)
    {
        int sample_idx = i * n_channels + ch;
        double y = wav_in[sample_idx];

        double abs = fabs(y);
        if(abs >= clip_limit){
            if (y < 0){
                y = -clip_limit;
            }else{
                y = clip_limit;
            }
        }
        wav_out[sample_idx] = y;
    }
}